#include "hip/hip_runtime.h"

#include <stdio.h>
#include <wb.h>

#define BLOCK_SIZE 512
// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

#define FATAL(msg, ...) \
    do {\
        fprintf(stderr, "[%s:%d] ", msg, "\n", __FILE__, __LINE__, ##__VA_ARGS__);\
        exit(-1);\
    } while(0)

#if __BYTE_ORDER != __LITTLE_ENDIAN
# error "File I/O is not implemented for this system: wrong endianness."
#endif


// Global queuing stub
__global__ void gpu_global_queuing_kernel(
  int *nodePtrs,
  int *nodeNeighbors,
  int *nodeVisited,
  int *currLevelNodes,
  int *nextLevelNodes,
  const unsigned int numCurrLevelNodes,
  int *numNextLevelNodes) {


  //@@ Insert global queuing code here

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the global queue (already marked in the exchange)

}

// Block queuing stub
__global__ void gpu_block_queuing_kernel(
  int *nodePtrs,
  int *nodeNeighbors,
  int *nodeVisited,
  int *currLevelNodes,
  int *nextLevelNodes,
  const unsigned int numCurrLevelNodes,
  int *numNextLevelNodes) {

  //@@ Insert block queuing code here

  // Initialize shared memory queue
  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the block queue
  // If full, add it to the global queue
  // Allocate space for block queue to go into global queue
  // Store block queue in global queue

}

// Host function for global queuing invocation
void gpu_global_queuing(
  int *nodePtrs,
  int *nodeNeighbors,
  int *nodeVisited,
  int *currLevelNodes,
  int *nextLevelNodes,
  unsigned int numCurrLevelNodes,
  int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queuing_kernel <<< numBlocks , BLOCK_SIZE >>> (nodePtrs,
    nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
    numCurrLevelNodes, numNextLevelNodes);

}

// Host function for block queuing kernel invocation
void gpu_block_queuing(
  int *nodePtrs,
  int *nodeNeighbors,
  int *nodeVisited,
  int *currLevelNodes,
  int *nextLevelNodes,
  unsigned int numCurrLevelNodes,
  int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queuing_kernel <<< numBlocks , BLOCK_SIZE >>> (nodePtrs,
    nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
    numCurrLevelNodes, numNextLevelNodes);

}



int main(int argc, char* argv[])
{
    // Variables
    int numNodes;
    int *nodePtrs_h;
    int *nodeNeighbors_h;
    int *nodeVisited_h;
    int numTotalNeighbors_h;
    int *currLevelNodes_h;
    int *nextLevelNodes_h;
    int numCurrLevelNodes;
    int numNextLevelNodes_h;
    int *nodePtrs_d;
    int *nodeNeighbors_d;
    int *nodeVisited_d;
    int *currLevelNodes_d;
    int *nextLevelNodes_d;
    int *numNextLevelNodes_d;
    hipError_t cuda_ret;

    enum Mode {GPU_GLOBAL_QUEUE = 2, GPU_BLOCK_QUEUE};


    wbArg_t args = wbArg_read(argc, argv);
    Mode mode = (Mode)wbImport_flag(wbArg_getInputFile(args, 0));

    nodePtrs_h = (int *)wbImport(wbArg_getInputFile(args, 1), &numNodes, "Integer");
    nodeNeighbors_h = (int *)wbImport(wbArg_getInputFile(args, 2), &numTotalNeighbors_h, "Integer");
   

    nodeVisited_h = (int *)wbImport(wbArg_getInputFile(args, 3), &numNodes, "Integer");
    currLevelNodes_h = (int *)wbImport(wbArg_getInputFile(args, 4), &numCurrLevelNodes, "Integer");

    // (do not modify) Datasets should be consistent 
    assert(nodePtrs_h[numNodes] == numTotalNeighbors_h);

    // (do not modify) Prepare next level containers (i.e. output variables)
    numNextLevelNodes_h = 0;
    nextLevelNodes_h = (int*) malloc((numNodes)*sizeof(int));

    printf("# Mode = %u\n", mode);
    printf("# Nodes = %u\n", numNodes);
    printf("# Total Neighbors = %d\n", numTotalNeighbors_h);
    printf("# Current Level Nodes = %d\n", numCurrLevelNodes);

    // (do not modify) Allocate device variables --------------------------

    printf("Allocating device variables..."); fflush(stdout);

    cuda_ret = hipMalloc((void**)&nodePtrs_d,
      (numNodes + 1)*sizeof(int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&nodeVisited_d, numNodes*sizeof(int));
    if(cuda_ret!=hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&nodeNeighbors_d,
      nodePtrs_h[numNodes]*sizeof(int));
    if(cuda_ret!=hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&currLevelNodes_d,
      numCurrLevelNodes*sizeof(int));
    if(cuda_ret!=hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&numNextLevelNodes_d,
      sizeof(int));
    if(cuda_ret!=hipSuccess) FATAL("Unable to allocate device memory");

    cuda_ret = hipMalloc((void**)&nextLevelNodes_d,
      (numNodes)*sizeof(int));
    if(cuda_ret!=hipSuccess) FATAL("Unable to allocate device memory");

    hipDeviceSynchronize();

    // (do not modify) Copy host variables to device --------------------

    printf("Copying data from host to device..."); fflush(stdout);

    cuda_ret = hipMemcpy(nodePtrs_d, nodePtrs_h,
      (numNodes + 1)*sizeof(int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
      FATAL("Unable to copy memory to the device");
    }

    cuda_ret = hipMemcpy(nodeVisited_d, nodeVisited_h,
      numNodes*sizeof(int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
      FATAL("Unable to copy memory to the device");
    }

    cuda_ret = hipMemcpy(nodeNeighbors_d, nodeNeighbors_h,
      nodePtrs_h[numNodes]*sizeof(int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
      FATAL("Unable to copy memory to the device");
    }

    cuda_ret = hipMemcpy(currLevelNodes_d, currLevelNodes_h,
      numCurrLevelNodes*sizeof(int), hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) {
      FATAL("Unable to copy memory to the device");
    }

    cuda_ret = hipMemset(numNextLevelNodes_d, 0, sizeof(int));
    if(cuda_ret != hipSuccess) {
      FATAL("Unable to copy memory to the device");
    }

    hipDeviceSynchronize();

    // (do not modify) Launch kernel ----------------------------------------

    printf("Launching kernel ");

    if(mode == GPU_GLOBAL_QUEUE) {
        printf("(GPU with global queuing)...");fflush(stdout);
        gpu_global_queuing(nodePtrs_d, nodeNeighbors_d, nodeVisited_d,
          currLevelNodes_d, nextLevelNodes_d, numCurrLevelNodes,
          numNextLevelNodes_d);
        cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");
    } else if(mode == GPU_BLOCK_QUEUE) {
        printf("(GPU with block and global queuing)...");fflush(stdout);
        gpu_block_queuing(nodePtrs_d, nodeNeighbors_d, nodeVisited_d,
          currLevelNodes_d, nextLevelNodes_d, numCurrLevelNodes,
          numNextLevelNodes_d);
        cuda_ret = hipDeviceSynchronize();
        if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");
    } else {
        printf("Invalid mode!\n");
        exit(0);
    }


    // (do not modify) Copy device variables from host ----------------------

    printf("Copying data from device to host..."); fflush(stdout);

    cuda_ret = hipMemcpy(&numNextLevelNodes_h, numNextLevelNodes_d,
      sizeof(int), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

    cuda_ret = hipMemcpy(nextLevelNodes_h, nextLevelNodes_d,
      numNodes*sizeof(int), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

    cuda_ret = hipMemcpy(nodeVisited_h, nodeVisited_d,
      numNodes*sizeof(int), hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

    hipDeviceSynchronize();


    // (do not modify) Verify correctness -------------------------------------
    // Only check that the visited nodes match the reference implementation

    wbSolution(args, nodeVisited_h, numNodes);

    // (do not modify) Free memory ------------------------------------------------------------
    free(nodePtrs_h);
    free(nodeVisited_h);
    free(nodeNeighbors_h);
    free(currLevelNodes_h);
    free(nextLevelNodes_h);
    hipFree(nodePtrs_d);
    hipFree(nodeVisited_d);
    hipFree(nodeNeighbors_d);
    hipFree(currLevelNodes_d);
    hipFree(numNextLevelNodes_d);
    hipFree(nextLevelNodes_d);

    return 0;
}

